#include "GpuManager.h"
#include <iostream>
#include <hipfft/hipfft.h>


static hipfftHandle fftplan;

GpuManager::GpuManager()
{
  // initialize Context
  hipGetDeviceCount(&num_devices_);
  std::cout << "Found " << num_devices_ << " GPUs" << std::endl;

  checkCudaErrors(hipDeviceSynchronize());
  
  // init cufft and cuda context by creating a plan, this is completely unused, but just forces handle creation here.
  int dims[] = {int(256), int(256)};
  size_t workSize;
  checkCudaErrors(hipfftCreate(&fftplan));
  checkCudaErrors(hipfftMakePlanMany(
        fftplan, 2, dims, 0, 0, 0, 0, 0, 0, HIPFFT_C2C, 10, &workSize));
}

GpuManager::~GpuManager() {
    hipfftDestroy(fftplan);
    hipDeviceReset();
}


// instatiate the object
GpuManager gpuManager;
