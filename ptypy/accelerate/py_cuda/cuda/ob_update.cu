#include "hip/hip_runtime.h"
#include <iostream>
#include <utility>
#include <thrust/complex.h>
using thrust::complex;
__device__ inline void atomicAdd(complex<float>* x, complex<float> y)
    {
      float* xf = reinterpret_cast<float*>(x);
      atomicAdd(xf, y.real());
      atomicAdd(xf + 1, y.imag());
    }

extern "C"{
__global__ void ob_update(
    const complex<float>* exit_wave,
    int A,
    int B,
    int C,
    const complex<float>* __restrict__ probe,
    int D,
    int E,
    int F,
    complex<float>* obj,
    int G,
    int H,
    int I,
    const int* addr,
    complex<float>* denominator
    )
    {
      int bid = blockIdx.x;
      int tx = threadIdx.x;
      int ty = threadIdx.y;
      int addr_stride = 15;

      const int* oa = addr + 3 + bid * addr_stride;
      const int* pa = addr + bid * addr_stride;
      const int* ea = addr + 6 + bid * addr_stride;

      probe += pa[0] * E * F + pa[1] * F + pa[2];
      obj += oa[0] * H * I + oa[1] * I + oa[2];
      denominator += oa[0] * H * I + oa[1] * I + oa[2];

      assert(oa[0] * H * I + oa[1] * I + oa[2] + (B - 1) * I + C - 1 < G * H * I);

      exit_wave += ea[0] * B * C;

      for (int b = tx; b < B; b += blockDim.x)
      {
        for (int c = ty; c < C; c += blockDim.y)
        {
          atomicAdd(&obj[b * I + c], conj(probe[b * F + c]) * exit_wave[b * C + c] );
          atomicAdd(&denominator[b * I + c], probe[b * F + c] * conj(probe[b * F + c]) );
          }
       }
}

}